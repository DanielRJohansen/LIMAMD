

#include "Engine.cuh"
#include "Utilities.h"
#include "EngineUtils.cuh"
#include "Neighborlists.cuh"

#include "BoundaryCondition.cuh"
#include "EngineBodies.cuh"
#include "SimulationDevice.cuh"

#include <algorithm>






Engine::Engine(std::unique_ptr<Simulation> sim, BoundaryConditionSelect bc, std::unique_ptr<LimaLogger> logger)
	: bc_select(bc), m_logger(std::move(logger))
{
	simulation = std::move(sim);

	verifyEngine();

	//const int compound_size = sizeof(CompoundCompact);
	//const int nlsit_size = sizeof(NeighborList);
	//const int sssize = (sizeof(Float3) + sizeof(float)) * THREADS_PER_COMPOUNDBLOCK;
	//const int Ckernel_shared_mem = sizeof(CompoundCompact) + sizeof(NeighborList) +
	//	(2* sizeof(Float3)) * THREADS_PER_COMPOUNDBLOCK + sizeof(Coord) + sizeof(Float3) + clj_utilitybuffer_bytes;
	//static_assert(Ckernel_shared_mem < 45000, "Not enough shared memory for CompoundKernel");

	//const int sbsize = sizeof(SolventBlock);
	//const int Skernel_shared_mem = (sizeof(Float3) + 1) * SolventBlock::MAX_SOLVENTS_IN_BLOCK + sizeof(SolventBlock)
	//	+ sizeof(SolventTransferqueue<SolventBlockTransfermodule::max_queue_size>) * 6
	//	+ 4 + 4 * 3 * 2;

	// Create the Sim_dev
	if (sim_dev != nullptr) { throw std::runtime_error("Expected simdev to be null to move sim to device"); }
	sim_dev = new SimulationDevice(simulation->simparams_host, std::move(simulation->box_host));
	sim_dev = genericMoveToDevice(sim_dev, 1);




	//this->forcefield_host = forcefield_host;
	setDeviceConstantMemory();

	LIMA_UTILS::genericErrorCheck("Error during bootstrapTrajbufferWithCoords");

	// To create the NLists we need to bootstrap the traj_buffer, since it has no data yet
	bootstrapTrajbufferWithCoords();

	NeighborLists::updateNlists(sim_dev, simulation->simparams_host.bc_select, simulation->boxparams_host, timings.nlist);
	m_logger->finishSection("Engine Ready");
}

Engine::~Engine() {
	if (sim_dev != nullptr) {
		sim_dev->deleteMembers();
		hipFree(sim_dev);
	}

	assert(simulation == nullptr);
}

std::unique_ptr<Simulation> Engine::takeBackSim() {
	assert(sim_dev);
	simulation->box_host = SimUtils::copyToHost(sim_dev->box);
	return std::move(simulation);
}

void Engine::verifyEngine() {
	LIMA_UTILS::genericErrorCheck("Error before engine initialization.\n");

	if (simulation->boxparams_host.dims.x != BOX_LEN_NM) {
		//throw std::runtime_error(std::format("This simulations box_size of {} did not match the size the engine is compiled with {}", simulation->boxparams_host.dims.x, BOX_LEN_NM));
		throw std::runtime_error("This simulations box_size of "+ std::to_string(simulation->boxparams_host.dims.x)
		+ "did not match the size the engine is compiled with" + std::to_string(BOX_LEN_NM));
	}
}

void Engine::step() {
	LIMA_UTILS::genericErrorCheck("Error before step!");

	deviceMaster();	// Device first, otherwise offloading data always needs the last datapoint!
	//simulation->incStep();
	assert(simulation);
	assert(sim_dev);
	simulation->simsignals_host.step++;
	sim_dev->signals->step++;	// UNSAFE

	hostMaster();

	LIMA_UTILS::genericErrorCheck("Error after step!");
}

void Engine::hostMaster() {						// This is and MUST ALWAYS be called after the deviceMaster, and AFTER incStep()!
	auto t0 = std::chrono::high_resolution_clock::now();
	if ((simulation->getStep() % STEPS_PER_LOGTRANSFER) == 0) {
		offloadLoggingData();
		offloadTrajectory();


		if ((simulation->getStep() % STEPS_PER_THERMOSTAT) == 0 && ENABLE_BOXTEMP) {
			handleBoxtemp();
		}
		if (simulation->simparams_host.em_variant) {
			sim_dev->signals->thermostat_scalar = 0.9995f;	// The current implementation of em is only this, overwriting the velocity scaler to always be 0.5
		}

		//nlist_manager->handleNLISTS(simulation.get(), ALLOW_ASYNC_NLISTUPDATE, false, &timings.nlist);
		NeighborLists::updateNlists(sim_dev, simulation->simparams_host.bc_select, simulation->boxparams_host, timings.nlist);
	}
	if ((simulation->getStep() % STEPS_PER_TRAINDATATRANSFER) == 0) {
		offloadTrainData();
	}

	// Handle status
	runstatus.current_step = simulation->getStep();
	runstatus.critical_error_occured = sim_dev->signals->critical_error_encountered;	// TODO: Can i get this from simparams_host? UNSAFE
	// most recent positions are handled automaticall by transfer_traj
	runstatus.simulation_finished = runstatus.current_step >= simulation->simparams_host.n_steps || runstatus.critical_error_occured;

	//if ((simulation->getStep() % STEPS_PER_THERMOSTAT) == 1) {	// So this runs 1 step AFTER handleBoxtemp
	//	simulation->box->thermostat_scalar = 1.f;
	//}

	const auto t1 = std::chrono::high_resolution_clock::now();
	const int cpu_duration = (int)std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();
	timings.cpu_master += cpu_duration;
}

void Engine::terminateSimulation() {
	const auto steps_since_transfer = simulation->getStep() % STEPS_PER_LOGTRANSFER;
	if ((steps_since_transfer) > LOG_EVERY_N_STEPS) {
		offloadLoggingData(steps_since_transfer);
		offloadTrajectory(steps_since_transfer);
	}
}
#include <assert.h>

//--------------------------------------------------------------------------	CPU workload --------------------------------------------------------------//

void Engine::offloadLoggingData(const int steps_to_transfer) {
	assert(steps_to_transfer <= simulation->getStep());

	const int64_t startstep = simulation->getStep() - steps_to_transfer;
	const int64_t startindex = LIMALOGSYSTEM::getMostRecentDataentryIndex(startstep);
	const int64_t indices_to_transfer = LIMALOGSYSTEM::getNIndicesBetweenSteps(startstep, simulation->getStep());

	hipMemcpy(
		simulation->potE_buffer->getBufferAtIndex(startindex),
		//&simulation->potE_buffer[step_relative * simulation->boxparams_host.total_particles_upperbound],
		sim_dev->databuffers->potE_buffer, 
		sizeof(float) * simulation->boxparams_host.total_particles_upperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

	hipMemcpy(
		simulation->vel_buffer->getBufferAtIndex(startindex),
		sim_dev->databuffers->vel_buffer,
		sizeof(float) * simulation->boxparams_host.total_particles_upperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

#ifdef GENERATETRAINDATA
	hipMemcpy(	// THIS IS PROLLY WRONG NOW
		&simulation->loggingdata[startindex * 10],
		sim_dev->databuffers->outdata, 
		sizeof(float) * 10 * indices_to_transfer,
		hipMemcpyDeviceToHost);
#endif
	hipDeviceSynchronize();
}

void Engine::offloadTrajectory(const int steps_to_transfer) {
#ifndef DONTGENDATA

	const int64_t startstep = simulation->getStep() - steps_to_transfer;
	const int64_t startindex = LIMALOGSYSTEM::getMostRecentDataentryIndex(startstep);
	const int64_t indices_to_transfer = LIMALOGSYSTEM::getNIndicesBetweenSteps(startstep, simulation->getStep());

	hipMemcpy(
		//&simulation->traj_buffer[step_relative * simulation->total_particles_upperbound],
		simulation->traj_buffer->getBufferAtIndex(startindex),
		sim_dev->databuffers->traj_buffer,
		sizeof(Float3) * simulation->boxparams_host.total_particles_upperbound * indices_to_transfer,
		hipMemcpyDeviceToHost
	);

	hipDeviceSynchronize();
#endif
	step_at_last_traj_transfer = simulation->getStep();
	runstatus.most_recent_positions = simulation->traj_buffer->getBufferAtIndex(LIMALOGSYSTEM::getMostRecentDataentryIndex(simulation->getStep()-1));
}


void Engine::offloadTrainData() {
#ifdef GENERATETRAINDATA
	uint64_t values_per_step = N_DATAGAN_VALUES * MAX_COMPOUND_PARTICLES * simulation->boxparams_host.n_compounds;
	if (values_per_step == 0) {
		return;	// No data to transfer
	}

	uint64_t step_offset = (simulation->getStep() - STEPS_PER_TRAINDATATRANSFER) * values_per_step;	// fix max_compound to the actual count save LOTS of space!. Might need a file in simout that specifies cnt for loading in other programs...
	hipMemcpy(&simulation->trainingdata[step_offset], sim_dev->databuffers->data_GAN, sizeof(Float3) * values_per_step * STEPS_PER_TRAINDATATRANSFER, hipMemcpyDeviceToHost);
	LIMA_UTILS::genericErrorCheck("Cuda error during traindata offloading\n");
#endif
}


void Engine::bootstrapTrajbufferWithCoords() {
	if (simulation->simparams_host.n_steps == 0) return;
	LIMA_UTILS::genericErrorCheck("Error during bootstrapTrajbufferWithCoords");

	std::vector<CompoundCoords> compoundcoords_array(simulation->boxparams_host.n_compounds);
	auto error = hipMemcpy(compoundcoords_array.data(), sim_dev->box->coordarray_circular_queue, sizeof(CompoundCoords) * simulation->boxparams_host.n_compounds, hipMemcpyDeviceToHost);
	LIMA_UTILS::genericErrorCheck(error);
	

	// We need to bootstrap step-0 which is used for traj-buffer
	for (int compound_id = 0; compound_id < simulation->boxparams_host.n_compounds; compound_id++) {
		for (int particle_id = 0; particle_id < MAX_COMPOUND_PARTICLES; particle_id++) {

			const Float3 particle_abspos = LIMAPOSITIONSYSTEM::getAbsolutePositionNM(compoundcoords_array[compound_id].origo, compoundcoords_array[compound_id].rel_positions[particle_id]);
			simulation->traj_buffer->getCompoundparticleDatapointAtIndex(compound_id, particle_id, 0) = particle_abspos;
		}
	}

	LIMA_UTILS::genericErrorCheck("Error during bootstrapTrajbufferWithCoords");
}




//--------------------------------------------------------------------------	SIMULATION BEGINS HERE --------------------------------------------------------------//

void Engine::deviceMaster() {
	const auto t0 = std::chrono::high_resolution_clock::now();
	hipDeviceSynchronize();


	if (simulation->boxparams_host.n_compounds > 0) {
		LAUNCH_GENERIC_KERNEL(compoundLJKernel, simulation->boxparams_host.n_compounds, THREADS_PER_COMPOUNDBLOCK, bc_select, sim_dev);
		//compoundLJKernel<BoundaryCondition> << < simulation->boxparams_host.n_compounds, THREADS_PER_COMPOUNDBLOCK >> > (sim_dev);
	}

	hipDeviceSynchronize();

	if (simulation->boxparams_host.n_bridges > 0) {
		LAUNCH_GENERIC_KERNEL(compoundBridgeKernel, simulation->boxparams_host.n_bridges, MAX_PARTICLES_IN_BRIDGE, bc_select, sim_dev);
		//compoundBridgeKernel<BoundaryCondition> <<< simulation->boxparams_host.n_bridges, MAX_PARTICLES_IN_BRIDGE >> > (sim_dev);	// Must come before compoundKernel()
	}

	hipDeviceSynchronize();
	if (simulation->boxparams_host.n_compounds > 0) {
		LAUNCH_GENERIC_KERNEL(compoundBondsAndIntegrationKernel, simulation->boxparams_host.n_compounds, THREADS_PER_COMPOUNDBLOCK, bc_select, sim_dev);
		//compoundBondsAndIntegrationKernel<BoundaryCondition> << <simulation->boxparams_host.n_compounds, THREADS_PER_COMPOUNDBLOCK >> > (sim_dev);
	}
	LIMA_UTILS::genericErrorCheck("Error after compoundForceKernel");
	const auto t1 = std::chrono::high_resolution_clock::now();


#ifdef ENABLE_SOLVENTS
	if (simulation->boxparams_host.n_solvents > 0) {
		LAUNCH_GENERIC_KERNEL(solventForceKernel, SolventBlocksCircularQueue::blocks_per_grid, SolventBlock::MAX_SOLVENTS_IN_BLOCK, bc_select, sim_dev);
		//solventForceKernel<BoundaryCondition> << < SolventBlocksCircularQueue::blocks_per_grid, SolventBlock::MAX_SOLVENTS_IN_BLOCK>> > (sim_dev);


		hipDeviceSynchronize();
		LIMA_UTILS::genericErrorCheck("Error after solventForceKernel");
		if (SolventBlocksCircularQueue::isTransferStep(simulation->getStep())) {
			LAUNCH_GENERIC_KERNEL(solventTransferKernel, SolventBlocksCircularQueue::blocks_per_grid, SolventBlockTransfermodule::max_queue_size, bc_select, sim_dev);
			//solventTransferKernel<BoundaryCondition> << < SolventBlocksCircularQueue::blocks_per_grid, SolventBlockTransfermodule::max_queue_size >> > (sim_dev);
		}
	}
	hipDeviceSynchronize();
	LIMA_UTILS::genericErrorCheck("Error after solventTransferKernel");
#endif
	const auto t2 = std::chrono::high_resolution_clock::now();

	const int compounds_duration = (int)std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();
	const int solvents_duration = (int)std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();

	timings.compound_kernels += compounds_duration;
	timings.solvent_kernels += solvents_duration;
}
