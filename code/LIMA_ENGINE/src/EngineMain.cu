#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

__global__ void addArrays(int* a, int* b, int* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}


template<typename T>
T* genericMoveToDevice(T* data_ptr, int n_elements) {	// Currently uses MallocManaged, switch to unmanaged for safer operation
    if (n_elements == 0) { return nullptr; }

    T* gpu_ptr = nullptr;
    size_t bytesize = n_elements * sizeof(T);

    hipMallocManaged(&gpu_ptr, bytesize);

    auto cuda_status = hipMemcpy(gpu_ptr, data_ptr, bytesize, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        std::cout << "\nCuda error code: " << cuda_status << " - " << hipGetErrorString(cuda_status) << std::endl;
        throw std::runtime_error("Move to device failed");
    }

    if (n_elements == 1)
        delete data_ptr;
    else
        delete[] data_ptr;

    return gpu_ptr;
}

bool errorCheck() {
    hipDeviceSynchronize();
    auto cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        std::cout << "\nCuda error code: " << cuda_status << " - " << hipGetErrorString(cuda_status) << std::endl;
        return 1;
    }
    return 0;
}

int main() {
    const int size = 256;
    int* a = new int[size];
    int* b = new int[size];
    int* c = new int[size];

    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    a = genericMoveToDevice(a, size);
    b = genericMoveToDevice(b, size);
    c = genericMoveToDevice(c, size);

    addArrays << <256 / 32, 32 >> > (a, b, c, size);
    if (errorCheck())
        return 1;

    for (int i = 0; i < size; i++) {
        if (c[i] != i * 2) {
            std::printf("Failed CUDA test. %d+%d != %d\n", a[i], b[i], c[i]);
            return 1;
        }
    }

    hipFree(a);
    hipFree(b);
    hipFree(c);
    std::printf("\tLIMA_ENGINE: CUDA-test successful\n");

    return 0;
}
