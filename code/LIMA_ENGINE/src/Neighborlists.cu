#include "hip/hip_runtime.h"

#include <algorithm>
//#include <execution>
#include <algorithm>
#include <unordered_set>
#include "Neighborlists.cuh"
#include "EngineUtils.cuh"
#include "Utilities.h"

// ------------------------------------------------------------------------------------------- PRIVATE HELPERS -------------------------------------------------------------------------------------------//


bool neighborWithinCutoff(const Float3* pos_a, const Float3* pos_b, const float cutoff_nm) {		// This is used for compounds with a confining_particle_sphere from key_particle BEFORE CUTOFF begins
	const float dist = EngineUtils::calcHyperDistNM(pos_a, pos_b);
	return dist < cutoff_nm;
}


NListDataCollection::NListDataCollection(Simulation* simulation) {
	compound_neighborlists.resize(MAX_COMPOUNDS);
	auto cuda_status = hipMemcpy(compound_neighborlists.data(), simulation->sim_dev->box->compound_neighborlists, sizeof(NeighborList) * simulation->boxparams_host.n_compounds, hipMemcpyDeviceToHost);
	LIMA_UTILS::genericErrorCheck(cuda_status);

	compoundgrid = std::make_unique<CompoundGrid>();
}

void NListDataCollection::preparePositionData(const Simulation& simulation, const uint32_t step_at_update) {
	// Data for the current step has not yet been generated so we need to use the previous step.
	// For the very first step, engine has cheated and already written the traj from the initial setup.	
	const auto step = step_at_update == 0 ? 0 : step_at_update - 1;	
	const auto entryindex = LIMALOGSYSTEM::getMostRecentDataentryIndex(step);

	for (int compound_id = 0; compound_id < simulation.boxparams_host.n_compounds; compound_id++) {
		const int key_index = simulation.compounds_host[compound_id].key_particle_index;
		compound_key_positions[compound_id] = simulation.traj_buffer->getCompoundparticleDatapointAtIndex(compound_id, key_index, entryindex);
	}
}



namespace NListUtils {

	void matchCompoundNeighbors(Simulation* simulation, NListDataCollection* nlist_data_collection) 
	{
		for (int i = 0; i < simulation->boxparams_host.n_compounds; i++) {
			nlist_data_collection->compound_neighborlists[i].n_compound_neighbors = 0;
		}

		for (uint16_t id_self = 0; id_self < simulation->boxparams_host.n_compounds; id_self++) {

			NeighborList* nlist_self = &nlist_data_collection->compound_neighborlists[id_self];
			const float cutoff_add_self = simulation->compounds_host[id_self].radius;
			const Float3& pos_self = nlist_data_collection->compound_key_positions[id_self];	// abs pos [nm]


			// Go through all compounds in box, with higher ID than self!
			for (uint16_t id_other = id_self + 1; id_other < simulation->boxparams_host.n_compounds; id_other++) {	// For finding new nearby compounds, it is faster and simpler to just check all compounds, since there are so few
				NeighborList* nlist_candidate = &nlist_data_collection->compound_neighborlists[id_other];
				const Float3& pos_other = nlist_data_collection->compound_key_positions[id_other];
				const float cutoff_add_candidate = simulation->compounds_host[id_other].radius;
				
				if (neighborWithinCutoff(&pos_self, &pos_other, CUTOFF_NM + cutoff_add_self + cutoff_add_candidate)) {
					nlist_self->addCompound(id_other);
					nlist_candidate->addCompound(id_self);					
				}
			}
		}
	}

	void assignNearbyCompoundsToGridnodes(Simulation* simulation, NListDataCollection* nlist_data_collection) {
#ifdef ENABLE_SOLVENTS
		for (int compound_id = 0; compound_id < simulation->boxparams_host.n_compounds; compound_id++) 
		{
			nlist_data_collection->compound_neighborlists[compound_id].n_gridnodes = 0;

			const Float3& compound_pos = nlist_data_collection->compound_key_positions[compound_id];
			const NodeIndex& compound_nodeindex = LIMAPOSITIONSYSTEM::absolutePositionToNodeIndex(compound_pos);

			const float compound_radius = simulation->compounds_host[compound_id].radius;
			const float max_dist_nm = CUTOFF_NM + compound_radius;

			for (int x = -GRIDNODE_QUERY_RANGE; x <= GRIDNODE_QUERY_RANGE; x++) {
				for (int y = -GRIDNODE_QUERY_RANGE; y <= GRIDNODE_QUERY_RANGE; y++) {
					for (int z = -GRIDNODE_QUERY_RANGE; z <= GRIDNODE_QUERY_RANGE; z++) {

						NodeIndex query_origo = compound_nodeindex + NodeIndex{ x,y,z };
						LIMAPOSITIONSYSTEM::applyPBC(query_origo);

						CompoundGridNode* querynode = nlist_data_collection->compoundgrid->getBlockPtr(query_origo);
						const int querynode_id = CompoundGrid::get1dIndex(query_origo);

						const Float3 querynode_pos = LIMAPOSITIONSYSTEM::nodeIndexToAbsolutePosition(query_origo);
						const float dist = EngineUtils::calcHyperDistNM(&compound_pos, &querynode_pos);

						if (dist < max_dist_nm) {
							querynode->addNearbyCompound(compound_id);	// Add compound so solvents can see it
							nlist_data_collection->compound_neighborlists[compound_id].addGridnode(querynode_id);	// Add grid so compound can see solvents
						}						
					}
				}
			}
		}
#endif
	}

	// Important: do NOT call getStep during this funciton, as it runs async!!!!
	// This is a thread worker-function, so it can't own the object, thus i pass a ref to the engine object..
	void updateNeighborLists(Simulation* simulation, NListDataCollection* nlist_data_collection, volatile bool* finished, int* timing, std::mutex& mutex, const uint32_t step_at_update) {
		try {
			auto t0 = std::chrono::high_resolution_clock::now();
			mutex.lock();

			// Make key positions addressable in arrays: compound_key_positions and solvent_positions
			nlist_data_collection->preparePositionData(*simulation, step_at_update);

			// First do culling of neighbors that has left CUTOFF
			//NListUtils::cullDistantNeighbors(simulation, nlist_data_collection);

			// Add all compound->compound neighbors
			matchCompoundNeighbors(simulation, nlist_data_collection);

			// updateCompoundGrid
			nlist_data_collection->compoundgrid = std::make_unique<CompoundGrid>();	// Reset the grid. Maybe there is a way to do this faster?
			assignNearbyCompoundsToGridnodes(simulation, nlist_data_collection);

			auto t1 = std::chrono::high_resolution_clock::now();
			*timing = (int)std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();

			// SIGNALING MAIN THREAD //
			*finished = 1;		// Thread terminates here!
			mutex.unlock();	// Unlock
		}
		catch (const std::runtime_error& ex) {
			std::cerr << "\nCaught exception: " << ex.what() << std::endl;	// TODO: Remove before final release
		}
	}
}








// ------------------------------------------------------------------------------------------- PUBLIC INTERFACE -------------------------------------------------------------------------------------------//

NListManager::NListManager(Simulation* simulation) {
	nlist_data_collection = std::make_unique<NListDataCollection>(simulation);

	for (int i = 0; i < simulation->boxparams_host.n_compounds; i++) {
		nlist_data_collection->compound_neighborlists[i].associated_id = i;
	}
}

// Main sim thread enters this block, so make sure it can leave VERY quickly
void NListManager::handleNLISTS(Simulation* simulation, const bool async, const bool force_update, int* timing) {
	const auto step = simulation->getStep();

	// Check whether we are getting too far behind
	if (stepsSinceUpdate(step) / STEPS_PER_NLIST_UPDATE > 4) {
#ifndef LIMAPUSH
		printf("\nWe are now %d nlist updates behind!\n", stepsSinceUpdate(step) / STEPS_PER_NLIST_UPDATE);
#endif
		//exit(1);
	}

	// If module is busy, return
	if (!m_mutex.try_lock()) { return; }

	// If new data is ready, push it
	if (updated_neighborlists_ready) {
		pushNlistsToDevice(simulation);
	}
	m_mutex.unlock();


	// If we dont need to update nlist, return
	if (!(stepsSinceUpdate(step) >= STEPS_PER_NLIST_UPDATE || step == 0)) { 
		return; 
	}

	if (async && !force_update) {
		std::thread nlist_worker(NListUtils::updateNeighborLists, simulation, nlist_data_collection.get(), &updated_neighborlists_ready, timing, std::ref(m_mutex), step);
		nlist_worker.detach();
	}
	else {
		NListUtils::updateNeighborLists(simulation, nlist_data_collection.get(), &updated_neighborlists_ready, timing, m_mutex, step);
	}
	prev_update_step = step;

	// If we are not async we can update immediately. If force_update, we need to wait. In either case lock
	if (!async || force_update) {
		const std::chrono::microseconds sleep_duration{ 500 };
		while (!updated_neighborlists_ready) { std::this_thread::sleep_for(sleep_duration); }
		m_mutex.lock();
		pushNlistsToDevice(simulation);
		m_mutex.unlock();
	}
}


void NListManager::pushNlistsToDevice(Simulation* simulation) {
	hipMemcpy(simulation->sim_dev->box->compound_neighborlists, nlist_data_collection->compound_neighborlists.data(), sizeof(NeighborList) * simulation->boxparams_host.n_compounds, hipMemcpyHostToDevice);
	LIMA_UTILS::genericErrorCheck("Error after transferring compound neighborlists to device");

	//hipMemcpy(simulation->box->solvent_neighborlists, nlist_data_collection->solvent_neighborlists, sizeof(NeighborList) * simulation->n_solvents, hipMemcpyHostToDevice);

	hipMemcpy(simulation->sim_dev->box->compound_grid, nlist_data_collection->compoundgrid.get(), sizeof(CompoundGrid), hipMemcpyHostToDevice);
	LIMA_UTILS::genericErrorCheck("Error after transferring CompoundGrid to device");

	updated_neighborlists_ready = 0;
}
